
#include <opencv2/core/core.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"
#include "helperfunctions.h"

//using namespace cv;
//using namespace std;

int main(int argc, char** argv)
{
	if (argc == 2 && strcmp("help", argv[1]) == 0) {
		printHelpMessage(stdout);
		return 0;
	}
	//Megn�zz�k, hogy van-e megfelel� GPU
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		fprintf(stderr, "You don't have a CUDA enabled GPU. Buy one! Sorry.\n");
		return NO_DEVICE_ERROR;
	}
	hipSetDevice(0);	//TODO:  csekkolni a hib�t!!!

	float sigma_s, sigma_r;
	int r, threads;

	int returnValue = readConfigParameters(argc, argv, sigma_s, sigma_r, r, threads);
	if (returnValue != 0) {
		return returnValue;
	}

	cv::Mat image;
	image = cv::imread(argv[1], 0);		//beolvassuk a k�pet, 8 bit sz�rke�rnyalatoss� konvert�ljuk
	if (!image.data) {
		fprintf(stderr, "Could not open or find the input image\n\n");
		return NO_IMAGE_ERROR;
	}

	int width = image.cols;
	int height = image.rows;
	int imageSize = width * height;
	int spatialKernelSize = (2 * r + 1)*(2 * r + 1);
	int rangeKernelSize = 511;

	float *d_spatialKernel = NULL, *d_rangeKernel = NULL;
	unsigned char *d_inputImage = NULL, *d_outputImage = NULL;

	if (!doAllMallocs(d_spatialKernel, d_rangeKernel, d_inputImage, d_outputImage, spatialKernelSize, rangeKernelSize, imageSize)) {
		fprintf(stderr, "hipMalloc failed!\n\n");
		return CUDA_MALLOC_ERROR;
	}

	createSpatialKernel<<<1, spatialKernelSize>>>(d_spatialKernel, r, sigma_s);
	createRangeKernel<<<1, 511>>>(d_rangeKernel, sigma_r);

	hipMemcpy(d_inputImage, image.data, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

	int blocksX = (width + threads - 1) / threads;
	int blocksY = (height + threads - 1) / threads;

	int sharedMemSize = (spatialKernelSize + rangeKernelSize) * sizeof(float);

	bilateralFilter<<<dim3(blocksX, blocksY), dim3(threads, threads), sharedMemSize >>>
		(d_inputImage, d_outputImage, d_spatialKernel, d_rangeKernel, r, width, height);

	hipMemcpy(image.data, d_outputImage, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

	cv::imwrite(argv[2], image);

	hipFree(d_inputImage);
	hipFree(d_outputImage);
	hipFree(d_spatialKernel);
	hipFree(d_rangeKernel);
	return 0;
}