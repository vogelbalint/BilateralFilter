
#include "hip/hip_runtime.h"


#include "math.h"

#define MAX_RANGE_DIFF 255

//Gauss f�ggv�nyt sz�m�t. Fontos: x n�gyzet�t kell �tadni neki, meg a sigma-t.  
__device__ float gauss(float x_square, float sigma)
{
	return expf(- x_square / (2 * sigma * sigma));
}

//A t�rbeli kernelt el�re kisz�m�tom, hogy ne kelljen egy adott pixel eset�n a szumm�z�s egy adott l�p�s�ben exponenci�lis f�ggv�nyt
//sz�m�tani, mert ez dr�ga m�vlet. Egyszer�bb az, ha a lehets�ges �rt�keket kisz�m�tjuk, ezt betessz�k egy m�trixba (illetve egy t�mbbe)
//ezt a t�b�t bet�tj�k a shared mem�ri�ba is innen szedj�k majd el�.
//a spatialKernel t�mb tartalmazza a lehets�ges t�rbeli elt�r�sekhez tartoz� Gauss f�ggv�ny �rt�keket.
//r: a t�rbeli kernel sugara (vagyis k�t pixel k�z�tti legnagyobb t�rbeli elt�r�s, amit figyelembe vesz�nk, r).
//sigma: a sptial Gaiss f�ggv�nyhez tartot� digma.
__global__ void createSpatialKernel(float *spatialKernel, int r, float sigma)
{
	int n = 2 * r + 1;		//a kernel oldal�nak hossza
	int i = blockIdx.x - r;	//oszlop index a spatial kernelben
	int j = blockIdx.y - r;	//sor index a spatial kernelben
	float x_square = (float)(i * i + j * j);
	spatialKernel[blockIdx.x + n * blockIdx.y] = gauss(x_square, sigma);
}


//K�t pixel intenzit�s�nak k�l�nbs�ge 255*2+1 = 511 f�le �rt�k lehet (a legkisebb 0-255 = -255, a legnagyobb 255 - 0 = 255)
//�rdemes ezeket is el�re kisz�m�tani, mert egy adott pixelhez tartoz� G(I_i - I_j) (az inenzt�s k�l�nbs�ghez tartoz� Gauss)
//kisz�m�t�sa k�lts�ges m�velet, 511 pedig nem olyan nagy sz�m. Ez hasonl� az el�z� spatial kernelhez.
//a lehets�ges intenzit�s k�l�nbs�gekhez tartoz� Gauss �rt�keket t�rol� t�mb�t rangeKernel-nek nevezem (nem prec�z).
//az intenzit�s k�l�nbs�g abszol�t �rt�k�ek maximuma MAX_RANGE_DIFF
__global__ void createRangeKernel(float *rangeKernel, float sigma)
{
	//el�sz�r csak a pozit�v delte I -khez tartoz� Gausst sz�m�tjuk ki, mert szimmetrkus a f�ggv�ny
	int tid = threadIdx.x;
	if (tid >= MAX_RANGE_DIFF) {
		int deltaI = threadIdx.x - MAX_RANGE_DIFF;
		rangeKernel[tid] = gauss((float)(deltaI * deltaI), sigma);
	}

	__syncthreads();

	//�tm�soljuk a negat�v �rt�kekhez tartoz� cuccokat
	int last = MAX_RANGE_DIFF * 2;  //=510
	if (tid < MAX_RANGE_DIFF) {
		rangeKernel[tid] = rangeKernel[last - tid];
	}
}


//A bilaterel filtert megval�s�t� cuda kernel.
//es� k�t argumentum: a bemen� is kimen� k�p pixeleinek intenzit�s �rt�keit tartalmaz� t�mb�k
//spatialKernel, rangeKernel: a t�rbeli �s intenzit�sbeli k�l�nbs�gekhez tartoz� Gauss �rtkeket t�rol� t�mb�k.
//Ezekb�l sokszor olvasunk, ez�rt ezeket a shared mem�ri�ba m�soljuk.
//r: a spatial kernel sugara ; width, height: a k�p sz�less�ge �s magass�g, pixelben.
__global__ void bilateralFilter(unsigned char *in, unsigned char *out, float *spatialKernel, float *rangeKernel, int r,
								int width, int height)
{
	int n = 2 * r + 1;			//a spatial kernel oldal�nak hossza
	int spatialKernelSize = n * n;
	extern __shared__ float sharedData[];	//A shared memory t�rolja a spatial kernel �s a rangeKernel �rt�keit is, egym�s ut�n folytonosan 
	float *pSpatialKernel = &sharedData[r * n + r];					//a shared memory spatial kernelt t�rol� r�sz�nek k�zep�re mutat� ponter
	float *pRangeKernel = &sharedData[spatialKernelSize + 255];		//a shared memory range kernelt t�rol� r�sz�nek k�zep�re mutat

	//A shared memory felt�lt�se:
	//1. minden thread �tm�solja a megfelel� spatialKernel elemet
	int index = threadIdx.x + blockDim.x * threadIdx.y;
	int step = blockDim.x * blockDim.y;		//az �sszes thread sz�ma a blockban
	while (index < spatialKernelSize) {
		sharedData[index] = spatialKernel[index];
		index += step;
	}

	//2. minden thread �tm�solja a megfelel� rangeKernel elemet
	index = threadIdx.x + blockDim.x * threadIdx.y;
	int rangeKernelSize = 2 * MAX_RANGE_DIFF + 1;		//=511
	while (index < rangeKernelSize) {
		sharedData[index + spatialKernelSize] = rangeKernel[index];
		index += step;
	}

	__syncthreads();
	//megvagyunk a shared memory felt�lt�s�vel, j�het a l�nyeg:

	int x = threadIdx.x + blockIdx.x * blockDim.x;			//pixel koordin�t�k kisz�m�t�sa
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;			//a pixel intenzit�s�t t�rol� mem�ria indexe az in �s out t�mb�kben

	if (x < width && y < height) {				//csak az �rv�nyes pixeleket n�zz�k
		float summa = 0, weightSumma = 0;
		int intensity = in[offset];				//az adott pixel inenzit�sa

		for (int j = -r; j <= r; ++j) {			//j: sorindex
			int yj = y + j;						//az aktu�lisan vizsg�lt pixel y koordin�t�ja

			for (int i = -r; i <= r; ++i) {		//i: oszlopindex
				int xi = x + i;					//az aktu�lisan vizsg�lt pixel x koordin�t�ja

				if (xi >= 0 && xi < width && yj >= 0 && yj < height) {
					int offsetij = xi + yj * blockDim.x * gridDim.x;	//az xi , yj pixel intenzit�s�t t�rol� mem�ria indexe
					int intensityij = in[offsetij];						//az xi, yj pixel intenzit�sa
					int deltaI = intensityij - intensity;				//az intenzit�sok k�l�nbs�ge
					float temp = pSpatialKernel[i + j * n] * pRangeKernel[deltaI];
					weightSumma += temp;
					summa += temp * intensityij;
				}
			}
		}

		out[offset] = (unsigned char)(summa / weightSumma);		//TODO: ink�bb kerek�tsen, mint lev�gjon
	}
}


